#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define NUM_THREADS 1024
#define NUM_CHANNELS 1
#define DEBUG_IMG_IDX 20

typedef unsigned char uint8_t;

#include "stb_image.h"
#include "stb_image_write.h"
#include <stdint.h>

hipError_t contrastEnhancementCuda(uint8_t *img, uint8_t &min_host, uint8_t &max_host, const int size);

__global__ void minKernel(uint8_t* img, uint8_t* o_img, const int size) {

    // Shared memory for threads in the same block
    extern __shared__ uint8_t sdata_minKernel[];

    unsigned int tid = threadIdx.x;
    // TODO: Check blockDim * 2 option
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Initialize shared memory
    if (i + blockDim.x < size) {
        sdata_minKernel[tid] = img[i] < img[i + blockDim.x] ? img[i] : img[i + blockDim.x];
    }
    else {
        sdata_minKernel[tid] = img[i];
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s) {
            sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + s] ? sdata_minKernel[i] : sdata_minKernel[i + s];
        }
        __syncthreads();
    }
    
    // Unroll the last warp
    if (tid < 32) {
        sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + 32] ? sdata_minKernel[i] : sdata_minKernel[i + 32];
        sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + 16] ? sdata_minKernel[i] : sdata_minKernel[i + 16];
        sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + 8] ? sdata_minKernel[i] : sdata_minKernel[i + 8];
        sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + 4] ? sdata_minKernel[i] : sdata_minKernel[i + 4];
        sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + 2] ? sdata_minKernel[i] : sdata_minKernel[i + 2];
        sdata_minKernel[tid] = sdata_minKernel[i] < sdata_minKernel[i + 1] ? sdata_minKernel[i] : sdata_minKernel[i + 1];
    }

    if (tid == 0) {
        o_img[blockIdx.x] = sdata_minKernel[0];
    }
}


__global__ void maxKernel(uint8_t* img, uint8_t* o_img, const int size) {

    // Shared memory for threads in the same block
    extern __shared__ uint8_t sdata_maxKernel[];

    unsigned int tid = threadIdx.x;
    // TODO: Check blockDim * 2 option
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Initialize shared memory
    if (i + blockDim.x < size) {
        sdata_maxKernel[tid] = img[i] > img[i + blockDim.x] ? img[i] : img[i + blockDim.x];
    }
    else {
        sdata_maxKernel[tid] = img[i];
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s) {
            sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + s] ? sdata_maxKernel[i] : sdata_maxKernel[i + s];
        }
        __syncthreads();
    }

    // Unroll the last warp
    if (tid < 32) {
        sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + 32] ? sdata_maxKernel[i] : sdata_maxKernel[i + 32];
        sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + 16] ? sdata_maxKernel[i] : sdata_maxKernel[i + 16];
        sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + 8] ? sdata_maxKernel[i] : sdata_maxKernel[i + 8];
        sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + 4] ? sdata_maxKernel[i] : sdata_maxKernel[i + 4];
        sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + 2] ? sdata_maxKernel[i] : sdata_maxKernel[i + 2];
        sdata_maxKernel[tid] = sdata_maxKernel[i] > sdata_maxKernel[i + 1] ? sdata_maxKernel[i] : sdata_maxKernel[i + 1];
    }

    if (tid == 0) {
        o_img[blockIdx.x] = sdata_maxKernel[0];
    }
}


__global__ void subtractMinKernel(uint8_t* dev_img, uint8_t* dev_min, const int size) {

    // Shared min value for threads in the same block. No bank error since data is broadcasted.
    __shared__ uint8_t min_val;
        
    min_val = *dev_min;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        dev_img[i] = dev_img[i] - min_val;
    }
    __syncthreads();
}


__global__ void scaleKernel(uint8_t* dev_img, float* dev_scale, const int size) {

    // Shared min value for threads in the same block. No bank error since data is broadcasted.
    __shared__ float scale;

    scale = *dev_scale;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        // rintf rounds the float number to the closest integer 
        dev_img[i] = rintf(dev_img[i] * scale);
    }
    __syncthreads();
}


int main()
{
    int width; //image width
    int height; //image height
    int bpp;  //bytes per pixel if the image was RGB (not used)

    uint8_t min_host, max_host;
    
    // Load a grayscale bmp image to an unsigned integer array with its height and weight.
    //  (uint8_t is an alias for "unsigned char")
    uint8_t* image = stbi_load("./samples/640x426.bmp", &width, &height, &bpp, NUM_CHANNELS);

    // Print for sanity check
    printf("Bytes per pixel: %d \n", bpp / 3); //Image is grayscale, so bpp / 3;
    printf("Height: %d \n", height);
    printf("Width: %d \n", width);

    printf("\n### Orig val: %d \n", image[DEBUG_IMG_IDX]);

    hipError_t cudaStatus = contrastEnhancementCuda(image, min_host, max_host, width * height);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "contrastEnhancementCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // Write image array into a bmp file
    stbi_write_bmp("./out_img_640x426.bmp", width, height, 1, image);

    // Deallocate memory
    stbi_image_free(image);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t contrastEnhancementCuda(uint8_t *img, uint8_t &min_host, uint8_t &max_host, const int size)
{
    int blockSize = NUM_THREADS;
    int gridSize = size / blockSize + (size % blockSize != 0);

    dim3 grid, block;
    block.x = NUM_THREADS;
    grid.x = size / block.x + (size % block.x != 0);
    
    // Temp CPU array
    uint8_t* min_array = new uint8_t [grid.x];
    //uint8_t* min_array = (uint8_t*) malloc( sizeof(uint8_t) * (size / NUM_THREADS + (size % NUM_THREADS != 0))) ;

    // Device memory pointers
    uint8_t *dev_img = 0;
    uint8_t *dev_min_array = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

    // Allocate GPU memory for the image.
    cudaStatus = hipMalloc((void**)&dev_img, size * sizeof(uint8_t));
    //hipMemset(dev_img, 0, size * sizeof(uint8_t));

    // Allocate GPU memory for min values of separate blocks.
    cudaStatus = hipMalloc((void**)&dev_min_array, sizeof(uint8_t) * grid.x);
    //hipMemset(dev_min_array, 0, sizeof(uint8_t) * grid.x);

    // Copy the image from host memory to GPU.
    hipMemcpy(dev_img, img, size * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMempcy img failed!");
        goto Error;
    }
    
    int smem_size = block.x * sizeof(uint8_t);
 

    //// Launch a kernel on the GPU with one thread for each element.
    //minKernel<<<grid, block, smem_size, 0>>>(dev_img, dev_min_array, size);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "minMaxKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
   
    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(min_array, dev_min_array, sizeof(uint8_t) * grid.x, hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy min failed!");
    //    goto Error;
    //}
    //min_host = 255;
    //for (size_t i = 0; i < NUM_THREADS; i++)
    //{
    //    if (min_array[i] < min_host) {
    //        min_host = min_array[i];
    //    }
    //}
    //
    //maxKernel << <grid, block, smem_size, 0 >> > (dev_img, dev_min_array, size);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "minMaxKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(min_array, dev_min_array, sizeof(uint8_t) * grid.x, hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy max failed!");
    //    goto Error;
    //}

    //max_host = 0;
    //for (size_t i = 0; i < NUM_THREADS; i++)
    //{
    //    if (min_array[i] > max_host) {
    //        max_host = min_array[i];
    //    }
    //}
    //
    //printf("Min: %d \n Max: %d \n", min_host, max_host);
    
    // For debug purposes
    min_host= 10;
    max_host = 200;

    float scale_constant = 255.0f / (max_host - min_host);

    uint8_t* dev_min;
    float* dev_scale;

    cudaStatus = hipMalloc((void**)&dev_min, sizeof(uint8_t));
    cudaStatus = hipMemcpy(dev_min, &min_host, sizeof(uint8_t), hipMemcpyHostToDevice);
    subtractMinKernel<<<grid, block, sizeof(uint8_t), 0>>> (dev_img, dev_min, size);
    cudaStatus = hipMemcpy(img, dev_img, sizeof(uint8_t) * size, hipMemcpyDeviceToHost);
    printf("### After subtraction %d\n", img[DEBUG_IMG_IDX]);

    cudaStatus = hipMalloc((void**)&dev_scale, sizeof(float));
    cudaStatus = hipMemcpy(dev_scale, &scale_constant, sizeof(float), hipMemcpyHostToDevice);
    scaleKernel<< <grid, block, sizeof(float), 0 >> > (dev_img, dev_scale, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scaleKernel failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(img, dev_img, sizeof(uint8_t) * size, hipMemcpyDeviceToHost);
    printf("### After scaling %d, scale: %.3f\n", img[DEBUG_IMG_IDX], scale_constant);

Error:
    hipFree(dev_img);
    hipFree(dev_min_array);
    free(min_array);
    
    return cudaStatus;
}
